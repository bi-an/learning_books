#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <malloc.h>
#include <random>
//#include <hip/device_functions.h>
#include "../inc/MemFile.h"
#include "../inc/hip/hip_runtime_api.h"

#define BLOCK_SIZE 16
#define M 1024
#define N 2048
#define L 16

typedef struct{
	int width;
	int height;
	int stride;
	double* elements;
}Matrix;

__device__ __host__ double GetElement(const Matrix A, int row, int col) {
	return A.elements[row * A.stride + col];
}

__device__ __host__ void SetElement(Matrix A, int row, int col, double value) {
	A.elements[row * A.stride + col] = value;
}

__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
	Matrix Asub;
	Asub.width = BLOCK_SIZE;
	Asub.height = BLOCK_SIZE;
	Asub.stride = A.stride;
	Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
		+ BLOCK_SIZE * col];
	return Asub;
}

__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

void MatMul(const Matrix A, const Matrix B, Matrix C) {
	Matrix d_A;
	d_A.width = d_A.stride = A.stride, d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double);
	checkCudaErrors(hipMalloc(&d_A.elements, size));
	checkCudaErrors(hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice));

	Matrix d_B;
	d_B.width = d_B.stride = B.width, d_B.height = B.height;
	size = B.width * B.height * sizeof(double);
	checkCudaErrors(hipMalloc(&d_B.elements, size));
	checkCudaErrors(hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice));

	// Allocate result matrix
	Matrix d_C;
	d_C.width = d_C.stride = C.width, d_C.height = C.height;
	size = d_C.width * d_C.height * sizeof(double);
	checkCudaErrors(hipMalloc(&d_C.elements, size));

	// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((B.width + BLOCK_SIZE - 1) / BLOCK_SIZE, (A.height + BLOCK_SIZE - 1) / BLOCK_SIZE);
	MatMulKernel << <dimGrid, dimBlock >> > (d_A, d_B, d_C);

	checkCudaErrors(hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_A.elements));
	checkCudaErrors(hipFree(d_B.elements));
	checkCudaErrors(hipFree(d_C.elements));
}

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;

	Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
	double Cvalue = 0; // Ҫע�⣺����һ���߳��еı�����һ��Block�У�ʵ���� BLOCK_SIZE * BLOCK_SIZE ������

	int row = threadIdx.y;
	int col = threadIdx.x;

	for (int m = 0; m < ((A.width + BLOCK_SIZE - 1) / BLOCK_SIZE); m++) {
		Matrix Asub = GetSubMatrix(A, blockRow, m);
		Matrix Bsub = GetSubMatrix(B, m, blockCol);

		// �ù����ڴ��ݴ��ȫ���ڴ���ȡ��Asub��Bsub������ȫ���ڴ���ʴ���
		// �ô����Ϊ BLOCK_SIZE * BLOCK_SIZE * 2
		// ��Ȼ��A��һ�п��Զ���B�������У��ȵȣ��ܹ��ô����Ϊ (BLOCK_SIZE * BLOCK_SIZE)^2
		__shared__ double As[BLOCK_SIZE][BLOCK_SIZE]; // ������Ϊ block������ֻ�����һ�Σ��� for ѭ��û�й�ϵ
		__shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE]; // TODO: bank conflict

		As[row][col] = GetElement(Asub, row, col);
		Bs[row][col] = GetElement(Bsub, row, col);

		__syncthreads();

		for (int e = 0; e < BLOCK_SIZE; e++)
			Cvalue += As[row][e] * Bs[e][col];

		__syncthreads();
	}
	
	SetElement(Csub, row, col, Cvalue);
}

void SetMatrix(Matrix A, int m, int n) {
	for (int i = 0; i < m; i++)
		for (int j = 0; j < n; j++)
			SetElement(A, i, j, rand() % 10000 / 1e4);
}

int main() {
	Matrix A, B, C;
	A.width = A.stride = N, A.height = M;
	A.elements = (double*)malloc(M * N *  sizeof(double));
	B.width = B.stride = L, B.height = N;
	B.elements = (double *)malloc(N * L * sizeof(double));
	C.width = C.stride = L, C.height = M;
	C.elements = (double*)malloc(M*L * sizeof(double));

	SetMatrix(A, M, N);
	SetMatrix(B, N, L);

	MemFile::writeBin("A.dat", A.elements, N, M);
	MemFile::writeBin("B.dat", B.elements, L, N);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	MatMul(A, B, C);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("elapsed time: %f ms\n", elapsedTime); // ������Բ�׼��������Host-Device���ڴ濽����Device���ڴ��ͷ�ʱ�䡣

	MemFile::writeBin("C.dat", C.elements, L, M);

	return 0;
}